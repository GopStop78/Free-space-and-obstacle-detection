#include "hip/hip_runtime.h"
#include "opencv2/calib3d.hpp"
#include "opencv2/highgui.hpp"
#include <opencv2/imgproc.hpp>
#include <opencv2/contrib/contrib.hpp>
#include <opencv2/core/core.hpp>
#include "opencv2/gpu/gpu.hpp"
#include "opencv2/core/gpumat.hpp"
#include <iostream>
#include <string> 	
#include <stdio.h>
#include <fstream>
#include "Occupancy.h"
using namespace cv;
using namespace std;


__global__ void polarGridKernel(ushort *d_Depth,unsigned char* d_test,float d_Q13, float d_Q23,int d_width,int d_height)
{
	 int j = (blockIdx.x * blockDim.x) + threadIdx.x; //width
	 int i = (blockIdx.y * blockDim.y) + threadIdx.y; //height
	 i = d_height - i -1;
	 float y,z;
	  if((i==0) || (j==0) || (i==d_height-1) || (j==d_width-1))
 		     return;    
		
	if( (d_Depth[d_width*i + j]	 ==  65535) || (d_Depth[d_width*i + j] ==  0) ) //ignore any unassigned depth
        		return;

           	y = (((float)(i)+d_Q13) * (float)d_Depth[d_width*i + j]) / d_Q23 ; //yc
            z = (float)d_Depth[d_width*i + j];//f*b/d
            //k = j;
            ///m = (z - zMinVal)/gridSize;
             //y dimension in point cloud is inverted
            //900 -> 600 , 1800 -> 800
            if(y <=  THRESHOLD) //what is good threshold  ??
            {
            	//PolarGrid.at<char>(m,k) = 0; //obstcale
            	d_test[d_width*i + j]= 0;
            }
            else //if(PolarGrid.at<char>(m,k) != 0)
            {
            	//PolarGrid.at<char>(m,k) = 255;
            	d_test[d_width*i + j]= 255;
	        }


}
void calculatePointCloudPolarGridGPU(const cv::Mat& Depth, cv::Mat &test, const cv::Mat& Q, int gridSize, double heightPercentage)
{
    // Getting the interesting parameters from Q, everything else is zero or one
    int k,m;
    float x,y,z;
    double zMinVal , zMaxVal; 
    float split = 1 - heightPercentage;
    float Q03 = Q.at<float>(0, 3);
    float Q13 = Q.at<float>(1, 3);
    float Q23 = Q.at<float>(2, 3);
    float Q32 = Q.at<float>(3, 2);
    float Q33 = Q.at<float>(3, 3);
    minMaxLoc( Depth, &zMinVal, &zMaxVal );
    int grid_rows = (zMaxVal - zMinVal )/gridSize  + 1;
	int grid_cols = Depth.cols ;
	test  = cv::Mat::zeros( (Depth.rows) , Depth.cols ,  CV_8UC1); //black means empty
	//Mat temp = Mat::zeros(  Depth.rows - (Depth.rows)*heightPercentage , Depth.cols ,  CV_8UC1);
 	//Mat PolarGrid;


    //PolarGrid = cv::Mat::zeros( grid_rows , grid_cols ,  CV_8U); //black means empty
    //PolarGrid = Scalar(127);
    test = Scalar(127);
    ushort *d_Depth;
    unsigned char * d_test;
    hipMalloc((void**) &d_Depth, (Depth.rows)*(Depth.cols)*sizeof(ushort));
    hipMalloc((void**) &d_test, (Depth.rows)*(Depth.cols)*sizeof(unsigned char));
    hipMemcpy(d_Depth, Depth.data, (Depth.rows)*(Depth.cols)*sizeof(ushort), hipMemcpyHostToDevice); 
    hipMemcpy(d_test, test.data, (Depth.rows)*(Depth.cols)*sizeof(unsigned char), hipMemcpyHostToDevice); 

	dim3 threadsPerBlock(16,16,1);
	dim3 numBlocks( (Depth.cols)/16, (Depth.rows*heightPercentage)/16,1); 
	polarGridKernel<<<numBlocks , threadsPerBlock>>>(d_Depth,d_test,Q13,Q23,(Depth.cols),(Depth.rows));
	hipMemcpy(test.data, d_test,(Depth.rows)*(Depth.cols)*sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipFree(d_Depth);
	hipFree(d_test);
 	//Mat L1((Depth.rows), (Depth.cols),CV_8UC1,tito);
 	//imshow("Toto",L1);
 	//waitKey(0);
}